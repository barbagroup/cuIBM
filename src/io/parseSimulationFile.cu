#include "io.h"
#include <parameterDB.h>
#include <yaml-cpp/yaml.h>
#include <fstream>

namespace io
{

using std::string;

timeScheme timeSchemeFromString(string &s)
{
  if (s == "EULER_EXPLICIT")
    return EULER_EXPLICIT;
  else if (s == "EULER_IMPLICIT")
    return EULER_IMPLICIT;
  else if (s == "ADAMS_BASHFORTH_2")
    return ADAMS_BASHFORTH_2;
  else if (s == "RUNGE_KUTTA_3")
    return RUNGE_KUTTA_3;
  else if (s == "CRANK_NICOLSON")
    return CRANK_NICOLSON;
  else
    return EULER_EXPLICIT;
}

preconditionerType preconditionerTypeFromString(string &s)
{
  if (s == "NONE")
    return NONE;
  else if (s == "DIAGONAL")
    return DIAGONAL;
  else if (s == "SMOOTHED_AGGREGATION")
    return SMOOTHED_AGGREGATION;
  else
    return NONE;
}

ibmScheme ibmSchemeFromString(string &s)
{
  if (s == "NAVIER_STOKES")
    return NAVIER_STOKES;
  else if (s == "SAIKI_BIRINGEN")
    return SAIKI_BIRINGEN;
  else if (s == "DIRECT_FORCING")
    return DIRECT_FORCING;
  else if (s == "TAIRA_COLONIUS")
    return TAIRA_COLONIUS;
  else if (s == "FADLUN_ET_AL")
    return FADLUN_ET_AL;
  else
    return NAVIER_STOKES;
}

interpolationType interpolationTypeFromString(string &s)
{
  if (s == "CONSTANT")
    return CONSTANT;
  else if (s == "LINEAR")
    return LINEAR;
  else
    return LINEAR;
}

void parseSimulation(const YAML::Node &node, parameterDB &DB)
{
	real   dt = 0.02,
	       scaleCV = 2.0;
	int    startStep = 0,
	       nt = 100,
	       nsave = 100;
	string ibmSch = "NAVIER_STOKES",
	       convSch = "EULER_EXPLICIT",
	       diffSch = "EULER_IMPLICIT",
	       interpType = "LINEAR";

	// read simulation parameters
	node["dt"] >> dt;
	node["nsave"] >> nsave;
	node["nt"] >> nt;
	node["ibmScheme"] >> ibmSch;
	try
	{
		node["startStep"] >> startStep;
	}
	catch(...)
	{
	}
	try
	{
		node["timeScheme"][0] >> convSch;
		node["timeScheme"][1] >> diffSch;
	}
	catch(...)
	{		
	}
	try
	{
		node["scaleCV"] >> scaleCV;
	}
	catch(...)
	{
	}
	try
	{
		node["interpolationType"] >> interpType;
	}
	catch(...)
	{
	}

	// write to DB
	string dbKey = "simulation";
	DB[dbKey]["dt"].set<real>(dt);
	DB[dbKey]["scaleCV"].set<real>(scaleCV);
	DB[dbKey]["startStep"].set<int>(startStep);
	DB[dbKey]["nt"].set<int>(nt);
	DB[dbKey]["nsave"].set<int>(nsave);
	DB[dbKey]["ibmScheme"].set<ibmScheme>(ibmSchemeFromString(ibmSch));
	DB[dbKey]["convTimeScheme"].set<timeScheme>(timeSchemeFromString(convSch));
	DB[dbKey]["diffTimeScheme"].set<timeScheme>(timeSchemeFromString(diffSch));
	DB[dbKey]["interpolationType"].set<interpolationType>(interpolationTypeFromString(interpType));

	string system = "velocity", linearSolver = "CG", preconditioner = "DIAGONAL";
	real tol = 1e-5;
	int maxIter = 10000;

	const YAML::Node &solvers = node["linearSolvers"];
	for (unsigned int i=0; i<solvers.size(); i++)
	{
		// read linear solver options
		solvers[i]["system"] >> system;
		solvers[i]["solver"] >> linearSolver;
		try
		{
			solvers[i]["preconditioner"] >> preconditioner;
		}
		catch(...)
		{
		}
		try
		{
			solvers[i]["tolerance"] >> tol;
		}
		catch(...)
		{
		}
		try
		{
			solvers[i]["maxIterations"] >> maxIter;
		}
		catch(...)
		{
		}

		// write to DB
		string dbKey = system + "Solve";
		DB[dbKey]["solver"].set<string>(linearSolver);
		DB[dbKey]["preconditioner"].set<preconditionerType>(preconditionerTypeFromString(preconditioner));
		DB[dbKey]["tolerance"].set<real>(tol);
		DB[dbKey]["maxIterations"].set<int>(maxIter);
	}
}

void parseSimulationFile(std::string &simFile, parameterDB &DB)
{
	std::ifstream fin(simFile.c_str());
	YAML::Parser parser(fin);
	YAML::Node doc;
	parser.GetNextDocument(doc);

	for(unsigned int i=0; i<doc.size(); i++)
		parseSimulation(doc[i], DB);
}

} // end namespace io
